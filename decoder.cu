#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <regex.h>
#include <iostream>
#include <fstream>
#include <map>
#include <vector>
#include <string>
#include <queue>
#include <time.h>
#include <sstream>

using namespace std;

/* --------------------------------- Global Definitions ------------------------------------ */

// In order:  the number of plaintext letters, the number of ciphertext letters, and the
//            length of the cipher.

int plain_num;
int cipher_num;
long cipher_length;

// Character vectors to represent the plaintext and ciphertext letters.

// These are only used in setting up: in the actual algorithm, the first ciphertext letter
// is given the index 0, the second the index 1, and so on.  So if the ciphertext letters are
// '_', 'a', and 'b', then 'ab ba' would be transformed into '1, 2, 0, 2, 1'.
// This allows us to just use the ciphertext letters and plaintext letters as indices, without
// an extra read into memory.  It has proven to speed up the program.

vector<char> cipher_alpha(0);
vector<char> plain_alpha(0);

// These hold the cipher string (as said before, as numbers, not as characters).
// The vector is used in setting up, since we don't know the size of the cipher beforehand.
// Once the string is read and the vector is full, we allocate the array for all future use.
// This is done for speed.

vector<int> cipher_string_vec(0);
int * cipher_string;

// Arrays to hold the unigram, bigram, and trigram probabilities.
// They are all one - dimensional.  Probabilities are accessed as follows:
// Unigram: P(x) = *(unigram + x)
// Bigram: P(x | y) = *(bigram + x + y * plain_num)
// Trigram: P(x | zy) = *(trigram + x + y * plain_num + z * plain_num * plain_num)

float * unigram;
float * bigram;
float * trigram;

// Maps characters to the integers representations

map<char, int> inv_cipher;
map<char, int> inv_plain;

// These are CUDA variables (defined here so that both the main program and the
// CUDA device can see them).
// greenhouse and backpointers are the tables - greenhouse holds the probabilities, 
// and backpointers the back pointers.
// cuda(Uni|Bi|Tri)grams are the CUDA versions of the above unigram, bigram, and trigram
// arrays.

__device__ float * greenhouse;
__device__ int * backpointers;
__device__ float * cudaUnigrams;
__device__ float * cudaBigrams;
__device__ float * cudaTrigrams;

// This is just a bunch of stuff put in to make the CUDA part of the program work right.

texture<float, 1, hipReadModeNormalizedFloat> texRef;
hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
const textureReference* texRefPtr;

/* ------------------------------End Global Definitions ------------------------------------ */

/* ---------------------------------------- Classes ---------------------------------------- */

// This class is just a comparison function for the A* heap.  It looks as pairs of probailities
// and partial solutions, and orders them according to the probability.  On a tie, the larger 
// solution is given precedence.
// (Note: the probabilities are in the negative log domain, so will seem backwards.)

class pair_compare
{

public:
  bool operator() (const pair<float, map<int, int> > &lhs, const pair<float, map<int, int> > &rhs) const
  {
    return (lhs.first > rhs.first) || ((lhs.first == rhs.first) && (lhs.second.size() < rhs.second.size()));
  }
};

/* ------------------------------------ End Classes ---------------------------------------- */

/* ----------------------------------- CUDA Functions -------------------------------------- */

// These parts are the actual function calls to the CUDA architecture.  Each one fills one
// plaintext cell (k) of one row (l) of the table. The variables k and l are called in
// parallel, so they aren't part of the arguments: instead, they're taken from the number
// of the thread created. This takes up the first two lines of all of the functions.

// This fills the first column of the table.  It just gives every cell its corresponding 
// unigram probability.  It gives zero probability if the cell is inconsistant with a solution
// (i.e., if 'a' maps to 'b' in the partial solution, the cell for 'a' maps to 'c' goes to -1),
// or if it is in conflict with the path through that cell (i.e., if we are lookong at a table 
// indicting the probabilities for 'g' mapping to 'h' *at that point*, we cannot map 'g' to 'j',
// as well.)
// Note: the partial solution used is the given solution plus one extra letter, given in the 
// arguments and runin parallel.
// 
// Parameters:
//   - greenhouse: the array holding the probabilities for the paths.  
//                   Note (1): To save space, we only store 3 values on the 'i' dimension.
//                             We wrap around using i % 3.
//                   Note (2): It is one - dimensional: greenhouse[i, l, k] is taken as
//                             *(greenhouse + (((i) % 3) * plain_num + l) * plain_num + k).
//   -plain_num: the number of plaintext letters.
//   -b: the next ciphertext letter to be added.
//   -part_soln: an array holding the current guess of the partial solution.  It always holds
//               all of the plaintext letters, but the ones that are not yet set all point to -1.
//               (indices are plaintext letters, values are ciphertext)
//   -part_inv: same as above, but ciphertext to plain text.
//   -c: the cihertext letter at the beginning of the cipher (i.e., it the ciphertext is '5, 2, 5, 6'
//       after being tranformed into ints, c here would be '5').
//   -unigram: the unigram array.
//   -countFnCalls: a variable used to determine l and k after breaking the thread blocks into 
//                  manageable sizes.  Basically, it just lets the program be used on CUDA.
//                  we can only call a certain number of threads at once, so we call blocks of
//                  fixed sizes.
//                  This is the number of blocks that have been called.
//   -numK: same as above, but this is the size of the blocks.
//
// Returns: Nothing, but changes greenhouse (the CUDA version) in place.

__global__ void rowZero(float* greenhouse, int plain_num, int b, int * part_soln, int * part_inv, int c, float * unigram, int countFnCalls, int numK){
  int l = threadIdx.x;
  int k = __fadd_rn(threadIdx.y, __fmul_rn(countFnCalls, numK));
  if (k < plain_num){
    int temp = __fadd_rn(k, __fmul_rn(plain_num, l));
    // The following check is common to all CUDA functions.
    // (l == k) == (c == b) is true iff either both ciphertexts
    // and both plaintexts are the same, of if both are different.
    // The other four checks determine compatibility with the partial solution.

    if(((l == k) == (c == b)) 
       && ((*(part_soln + l) == -1) || (*(part_soln + l) == c))
       && ((*(part_inv + c) == -1) || (*(part_inv + c) == l)) 
       && ((*(part_soln + k) == -1) || (*(part_soln + k) == b))
       && ((*(part_inv + b) == -1) || (*(part_inv + b) == k)) ){
      *(greenhouse + temp) = *(unigram + l);
    } else {
      // In the log linear domain, -1 represents a zero probability.

      *(greenhouse + temp) = -1;
    }
  }
}

// This function is the same as above, but it fills the second column (index one).  It runs
// the same checks as before, and adds checks of the bigram probabilities.
//
// Parameters: 
//   same as rowZero, except for the following:
//   - backpointers: the array holding the backpointers.  The same notes apply as for the
//                   greenhouse array.
//   -bigram: the CUDA version of the bigram probabilities.
//
// Returns: Nothing, but changes the greenhouse and backpointers arrays in place.

__global__ void rowOne(float* greenhouse, int* backpointers, int plain_num, int b, int * part_soln, int * part_inv, int c, float * unigram, float * bigram, int countFnCalls, int numK){
  int l = threadIdx.x;
  int k = __fadd_rn(threadIdx.y, __fmul_rn(countFnCalls, numK));
  if (k < plain_num){
    int temp = __fadd_rn(k, __fmul_rn(plain_num, __fadd_rn(l, plain_num)));
    // Same as rowZero.
    // The last check is to ensure that there is a unigram count for the chosen 
    // plaintext (likely enough that the check may be unneeded).

    if(((l == k) == (c == b)) 
       && ((*(part_soln + l) == -1) || (*(part_soln + l) == c))
       && ((*(part_inv + c) == -1) || (*(part_inv + c) == l)) 
       && ((*(part_soln + k) == -1) || (*(part_soln + k) == b))
       && ((*(part_inv + b) == -1) || (*(part_inv + b) == k))
       && (*(unigram + l) > 0) ){
      int j;
      float best_prob = -1;
      int back = -1;
      int gHptr = k;
      int biPtr = l;
      // Loop over possible previous letters.

      for(j = 0; j < plain_num; j++){
        if((*(bigram + biPtr) > 0) && (*(greenhouse + gHptr) > 0)
           && ((best_prob < 0) 
               || (best_prob > *(bigram + biPtr) + *(greenhouse + gHptr)))){
              best_prob = *(bigram + biPtr) + *(greenhouse + gHptr);
              back = j;
        }
        gHptr = __fadd_rn(plain_num, gHptr);
        biPtr = __fadd_rn(plain_num, biPtr);
      }
        *(greenhouse + temp) = best_prob;
        *(backpointers + temp) = back;
      } else {
        *(greenhouse + temp) = -1;
        *(backpointers + temp) = -1;
    }
  }
}

// This function is the same as above, but it fills the third column (index two).  It runs
// the same checks as before, but adds trigram probability checks.  Also, it adds consistency
// checks on the letters: i.e., if the ciphertest is ' ..., 7, 9, 7, ... ', then both sevens
// are forced to be the same plaintext letters.  This window only lasts for two characters.
//
// Note: the loops and if statements are commented for readability.  I should probably have 
//       seperated this into smaller functions, but wanted to reduce the number of calls.
//       The loops are over indices j and j2: j is the plaintext of the previous letter
//       (index 1), and j2 is the second last plaintext letter (index 0).
//       The branches of the if statements are as follows:
//         if # 1: same as in rowZero.  Checks consistency of path and with partial solutions.
//                 If this doesn't pass, we assign zero probability.
//           if # 1.1 (1st branch of #1): Check if c1 (previous letter) is fixed in the solution.
//                                        If so, we only need to check the plaintext letter that
//                                        maps to it.
//             if # 1.1.1: Checks that the the value at the cell greenhouse(1, j, k) is nonzero
//                         (i.e., there is any path through that cell).  If this fails, we give zero
//                         probability, since this is the only path.
//               if # 1.1.1.1: Check to see if the ciphertext at c2 is in the partial solution.
//                             If it is, we only have to check one plaintext letter.  This is similar to
//                             check # 1.1.
//                 if # 1.1.1.1.1: A few checks here: Checks that there is a path through
//                                 greenhouse(0, j2, k), like in check 1.1.1, and checks the
//                                 bigram and trigram constraints.  Doesn't need to check the
//                                 '..., 7, 9, 7, ...' constraint, since this is covered by the partial 
//                                 solution.
//                                 If this fails, give zero probability.
//               else # 1.1.1.1: c2 is not known, so check all possible j2:
//                 if # 1.1.1.2.1 (second branch of fourth if): We're looping over j2s here, so do the same
//                                                              checks as in 1.1.1.1.1, but add the 
//                                                              '..., 7, 9, 7, ...' constraint, and check to
//                                                              see if the current path is both possible and
//                                                              beats anything we've checked before.  If it fails,
//                                                              we just go to the next j2.
//           else # 1.1: c1 is not known, so check all possible j:
//             if # 1.1.2: Check that there is a path through greenhouse(1, j, k), as in check #1.1.1.  Also, check
//                         the '..., 7, 9, 7, ...' constraint, and that the probability at greenhouse(1, j, k) is high enough
//                         to actually give us a better solution.  If this fails, go to the next j.
//               if # 1.1.2.1: Same as 1.1.1.1, different branch.
//                  if # 1.1.2.1.1: Same as 1.1.1.1.1, different branch.
//               else # 1.1.2.1: c2 is not known, just like else # 1.1.1.1.
//                  if # 1.1.2.2.1: Same as 1.1.1.2.1, different branch.
//
// Parameters: 
//   same as rowOne, except for the following:
//   - trigram: the CUDA array for the trigram probabilities.
//   - c1, c2: the cipher letters at the previous two indices (c1 is index 1, c2 is index 0).
//
// Returns: Nothing, but changes the greenhouse and backpointers arrays in place.

__global__ void rowTwo(float* greenhouse, int* backpointers, int plain_num, int b, int c, int c1, int c2, int * part_soln, int * part_inv, float * unigram, float * bigram, float * trigram, int countFnCalls, int numK){
  int l = threadIdx.x;
  int k = __fadd_rn(threadIdx.y, __fmul_rn(countFnCalls, numK));
   if (k < plain_num){
     int temp = __fadd_rn(k, __fmul_rn(plain_num, __fadd_rn(l, __fmul_rn(plain_num, 2))));
     if(((l == k) == (c == b)) // if #1
        && ((*(part_soln + l) == -1) || (*(part_soln + l) == c))
        && ((*(part_inv + c) == -1) || (*(part_inv + c) == l)) 
        && ((*(part_soln + k) == -1) || (*(part_soln + k) == b))
        && ((*(part_inv + b) == -1) || (*(part_inv + b) == k))
        && (*(unigram + l) > 0) ){
        int j;
        int j2;
        float best_prob = -1;
        int back = -1;
        if(*(part_inv + c1) >= 0){ // if #1.1
          j = *(part_inv + c1);
          if(*(greenhouse + (plain_num + j) * plain_num + k) > 0){ // if #1.1.1
            if(*(part_inv + c2) >= 0){ // if # 1.1.1.1
              j2 = *(part_inv + c2);
              if((*(greenhouse + j2 * plain_num + k) >= 0)
                 && (*(bigram + j + plain_num * j2) >= 0)
                 && (*(trigram + l + plain_num * (j + plain_num * j2)) >= 0)){ // if # 1.1.1.1.1
                best_prob = *(greenhouse + (plain_num + j) * plain_num + k) + *(trigram + l + plain_num * (j + plain_num * j2));
                back = j;
              } // if # 1.1.1.1.1
            } else { // if # 1.1.1.1
              for(j2 = 0; j2 < plain_num; j2++){ // for: index j2
                if((*(greenhouse + (plain_num + j2) * plain_num + k) >= 0)
                   && ((j2 == l) == (c == c2))
                   && ((j == j2) == (c1 == c2))
                   && (*(bigram + j + plain_num * j2) >= 0)
                   && (*(trigram + l + plain_num * (j + plain_num * j2)) >= 0) // if # 1.1.1.2.1
                   && ((best_prob < 0) 
                       || (best_prob > *(trigram + l + plain_num * (j + plain_num * j2)) + *(bigram + j + plain_num * j2) + *(greenhouse + j2 * plain_num + k)))){
                  best_prob = *(trigram + l + plain_num * (j + plain_num * j2)) + *(bigram + j + plain_num * j2) + *(greenhouse + j2 * plain_num + k);
                  back = j;
                } // if #1.1.1.2.1 
              } // for: index j2 
            } // if #1.1.1.1
          } // if # 1.1.1
        } else { // if # 1.1
           for(j = 0; j < plain_num; j++){ // for: index j
             if((*(greenhouse + (plain_num + j) * plain_num + k) >= 0)
                && ((j == l) == (c == c1))
                && ((best_prob < 0) || (*(greenhouse + (plain_num + j) * plain_num + k) < best_prob))){ // if #1.1.2
               if(*(part_inv + c2) >= 0){ // if # 1.1.2.1
                 j2 = *(part_inv + c2);
                 if((*(greenhouse + j2 * plain_num + k) >= 0)
                    && (*(bigram + j + plain_num * j2) >= 0)
                    && (*(trigram + l + plain_num * (j + plain_num * j2)) >= 0)){ // if # 1.1.2.1.1
                   best_prob = *(greenhouse + (plain_num + j) * plain_num + k) + *(trigram + l + plain_num * (j + plain_num * j2));
                   back = j;
                 } // if # 1.1.2.1.1
               } else { // if # 1.1.2.1
                 for(j2 = 0; j2 < plain_num; j2++){ // for: index j2
                   if((*(greenhouse + j2 * plain_num + k) >= 0)
                     && ((j2 == l) == (c == c2))
                     && ((j == j2) == (c1 == c2))
                     && (*(bigram + j + plain_num * j2) >= 0)
                     && (*(trigram + l + plain_num * (j + plain_num * j2)) >= 0) // if # 1.1.2.2.1
                     && ((best_prob < 0) 
                         || (best_prob > *(trigram + l + plain_num * (j + plain_num * j2)) + *(bigram + j + plain_num * j2) + *(greenhouse + j2 * plain_num + k)))){
                    best_prob = *(trigram + l + plain_num * (j + plain_num * j2)) + *(bigram + j + plain_num * j2) + *(greenhouse + j2 * plain_num + k);
                    back = j;
                  } // if #1.1.2.2.1 
                } // for: index j2 
              } // if #1.1.2.1
            } // if # 1.1.2
          } // for: index j
        } // if #1.1
      *(greenhouse + temp) = best_prob;
      *(backpointers + temp) = back;
    } else { // if # 1
      *(greenhouse + temp) = -1;
      *(backpointers + temp) = -1;
    } // if # 1
  }
}

// This function is the same as above, but it fills the every column column after the third (index > 2).  
// If is exactly the same as rowTwo, except now it starts to use the backpointer information.  Otherwise,
// the loops, checks, and arguments are almost exactly the same.  There are a few places where the pointers
// are fixed to save a bit of time, too.  Those have been done only on this function since the last three 
// functions are called once per run, while this one is called ~cipher_length times.
//
// Note: the loops and if statements are commented for readability.  I should probably have 
//       seperated this into smaller functions, but wanted to reduce the number of calls.
//       The loops are over indices j and j2: j is the plaintext of the previous letter
//       (index 1), and j2 is the second last plaintext letter (index 0).
//       The branches of the if statements are as follows:
//         if # 1: same as in rowZero.  Checks consistency of path and with partial solutions.
//                 If this doesn't pass, we assign zero probability.
//           if # 1.1 (1st branch of #1): Check if c1 (previous letter) is fixed in the solution.
//                                        If so, we only need to check the plaintext letter that
//                                        maps to it.
//             if # 1.1.1: Checks that the the value at the cell greenhouse(i - 1, j, k) is nonzero
//                         (i.e., there is any path through that cell).  If this fails, we give zero
//                         probability, since this is the only path.
//               if # 1.1.1.1: Check to see if the ciphertext at c2 is in the partial solution.
//                             If it is, we only have to check one plaintext letter.  This is similar to
//                             check # 1.1.
//                 if # 1.1.1.1.1: A few checks here: Checks that there is a path through
//                                 greenhouse(i - 2, j2, k), like in check 1.1.1, and checks the
//                                 bigram and trigram constraints.  Doesn't need to check the
//                                 '..., 7, 9, 7, ...' constraint, since this is covered by the partial 
//                                 solution.
//                                 If this fails, give zero probability.
//               else # 1.1.1.1: c2 is not known, so check all possible j2:
//                 if # 1.1.1.2.1 (second branch of fourth if): We're looping over j2s here, so do the same
//                                                              checks as in 1.1.1.1.1, but add the 
//                                                              '..., 7, 9, 7, ...' constraint, and check to
//                                                              see if the current path is both possible and
//                                                              beats anything we've checked before.  If it fails,
//                                                              we just go to the next j2.
//           else # 1.1: c1 is not known, so check all possible j:
//             if # 1.1.2: Check that there is a path through greenhouse(i - 1, j, k), as in check #1.1.1.  Also, check
//                         the '..., 7, 9, 7, ...' constraint, and that the probability at greenhouse(i - 1, j, k) is high 
//                         enough to actually give us a better solution.  If this fails, go to the next j.
//               if # 1.1.2.1: Same as 1.1.1.1, different branch.
//                  if # 1.1.2.1.1: Same as 1.1.1.1.1, different branch.
//               else # 1.1.2.1: c2 is not known, just like else # 1.1.1.1.
//                  if # 1.1.2.2.1: Same as 1.1.1.2.1, different branch.
//
// Parameters: 
//   same as rowTwo, except:
//   -i: the index of the current column.
//
// Returns: same as rowTwo.

__global__ void rowThreePlus(float* greenhouse, int* backpointers, int plain_num, int b, int c, int c1, int c2, int i, long cipher_length, int * part_soln, int * part_inv, float * unigram, float * bigram, float * trigram, int countFnCalls, int numK){
  int l = threadIdx.x;
  int k = __fadd_rn(threadIdx.y, __fmul_rn(countFnCalls, numK));
  int j;
  int j2;
  int back = -1;
  int step1;
  int triPtr;
  int lastPtr;
  float best_prob = -1;
  if (k < plain_num){
    int temp = __fadd_rn(k, __fmul_rn(plain_num, __fadd_rn(l, __fmul_rn(plain_num, i % 3))));
            if(((l == k) == (c == b)) // if #1
               && ((*(part_soln + l) == -1) || (*(part_soln + l) == c))
               && ((*(part_inv + c) == -1) || (*(part_inv + c) == l)) 
               && ((*(part_soln + k) == -1) || (*(part_soln + k) == b))
               && ((*(part_inv + b) == -1) || (*(part_inv + b) == k))
               && (*(unigram + l) > 0) ){
            step1 = plain_num * ((i - 1) % 3);
            if(*(part_inv + c1) >= 0){ // if #1.1
              j = *(part_inv + c1);
              lastPtr = __fadd_rn(__fmul_rn(__fadd_rn(step1, j), plain_num), k);
              if(*(greenhouse + lastPtr) > 0){ // if #1.1.1
                if(*(part_inv + c2) >= 0){ // if # 1.1.1.1
                  j2 = *(part_inv + c2);
                  triPtr = __fadd_rn(l, __fmul_rn(plain_num, (__fadd_rn(j, __fmul_rn(plain_num, j2)))));
                  if((*(greenhouse + (((i - 2) % 3) * plain_num + j2) * plain_num + k) >= 0)
                       && (*(trigram + triPtr) >= 0)){ // if # 1.1.1.1.1
                    best_prob = *(greenhouse + lastPtr) + *(trigram + triPtr);
                    back = j;
                  } // if # 1.1.1.1.1
                } else { // if # 1.1.1.1
                  for(j2 = 0; j2 < plain_num; j2++){ // for: index j2
                    if((*(greenhouse + (((i - 2) % 3) * plain_num + j2) * plain_num + k) >= 0)
                        && (*(backpointers + (((i - 2) % 3) * plain_num + j2) * plain_num + k) >= 0)
                        && ((j2 == l) == (c == c2))
                        && ((j == j2) == (c1 == c2))
                        && (*(trigram + l + plain_num * (j + plain_num * j2)) >= 0) // if # 1.1.1.2.1
                        && ((best_prob < 0) 
                              || (best_prob > *(trigram + l + plain_num * (j + plain_num * j2)) + *(trigram + j + plain_num * (j2 + plain_num * *(backpointers + (((i - 2) % 3) * plain_num + j2) * plain_num + k))) + *(greenhouse + (((i - 2) % 3) * plain_num + j2) * plain_num + k)))){
                        best_prob = *(trigram + l + plain_num * (j + plain_num * j2)) + *(trigram + j + plain_num * (j2 + plain_num * *(backpointers + (((i - 2) % 3) * plain_num + j2) * plain_num + k))) + *(greenhouse + (((i - 2) % 3) * plain_num + j2) * plain_num + k);
                        back = j;
                    } // if #1.1.1.2.1 
                  } // for: index j2 
                } // if #1.1.1.1
              } // if # 1.1.1
            } else { // if # 1.1
              for(j = 0; j < plain_num; j++){ // for: index j
                if((*(greenhouse + (((i - 1) % 3) * plain_num + j) * plain_num + k) >= 0)
                   && ((j == l) == (c == c1))
                   && ((best_prob < 0) || (*(greenhouse + (((i - 1) % 3) * plain_num + j) * plain_num + k) < best_prob))){ // if #1.1.2
                  if(*(part_inv + c2) >= 0){ // if # 1.1.2.1
                    j2 = *(part_inv + c2);
                    if((*(greenhouse + (((i - 2) % 3) * plain_num + j2) * plain_num + k) >= 0) // if # 1.1.2.1.1
                       && (*(trigram + l + plain_num * (j + plain_num * j2)) >= 0)
                       && ((best_prob < 0) || (best_prob > *(greenhouse + (((i - 1) % 3) * plain_num + j) * plain_num + k) + *(trigram + l + plain_num * (j + plain_num * j2))))){ 
                      best_prob = *(greenhouse + (((i - 1) % 3) * plain_num + j) * plain_num + k) + *(trigram + l + plain_num * (j + plain_num * j2));
                      back = j;
                    } // if # 1.1.2.1.1
                  } else { // if # 1.1.2.1
                    for(j2 = 0; j2 < plain_num; j2++){ // for: index j2
                      if((*(greenhouse + (((i - 2) % 3) * plain_num + j2) * plain_num + k) >= 0)
                        && (*(backpointers + (((i - 2) % 3) * plain_num + j2) * plain_num + k) >= 0)
                        && ((j2 == l) == (c == c2))
                        && ((j == j2) == (c1 == c2))
                        && (*(trigram + l + plain_num * (j + plain_num * j2)) >= 0) // if # 1.1.2.2.1
                        && ((best_prob < 0) 
                              || (best_prob > *(trigram + l + plain_num * (j + plain_num * j2)) + *(trigram + j + plain_num * (j2 + plain_num * *(backpointers + (((i - 2) % 3) * plain_num + j2) * plain_num + k))) + *(greenhouse + (((i - 2) % 3) * plain_num + j2) * plain_num + k)))){
                        best_prob = *(trigram + l + plain_num * (j + plain_num * j2)) + *(trigram + j + plain_num * (j2 + plain_num * *(backpointers + (((i - 2) % 3) * plain_num + j2) * plain_num + k))) + *(greenhouse + (((i - 2) % 3) * plain_num + j2) * plain_num + k);
                        back = j;
                      } // if #1.1.2.2.1 
                    } // for: index j2 
                  } // if #1.1.2.1
                } // if # 1.1.2
              } // for: index j
            } // if #1.1
            *(greenhouse + temp) = best_prob;
            *(backpointers + temp) = back;
          } else { // if # 1
            *(greenhouse + temp) = -1;
            *(backpointers + temp) = -1;
          } // if # 1
  }
}

/* --------------------------------End CUDA Functions -------------------------------------- */

/* -------------------------------- General Functions -------------------------------------- */

// This is the actual generalized Viterbi algorithm.  The conceptual details should already be known, so the details are:
//   (1) set up the block size to determine the number of threads per block.
//   (2) run the first column in parallel.
//   (3) run the second column in parallel.
//   (4) run the third column in parallel.
//   (4) succesively run the every remaining column in parallel.
//   (5) once finished, find the best probability over every remaining plaintext letter k,
//       where k points to the next ciphertext letter to be fixed.  This is done by looking at 
//       greenhouse(cipher_length - 1, l, k) for every possible l, and taking the best result.
//       put these into the result array.
//
//   The columns are treated differently for these reasons:
//     Column one can only use the unigram statistics.
//     Column two can only use bigram statistics.
//     Column three can't use back pointers.
//     The remaining columns can use everything.
//     I've seperated them into different functions, rather than different branches, so that the time taken to 
//     decide which branch to use is mininmized.
//
// Parameters:
//   -b: The next ciphertext letter to be fixed.
//   -plain_num: the number of plaintext letters.
//   -cipher_num: the number of ciphertext letters.
//   -part_soln: an array holding the current guess of the partial solution.  It always holds
//               all of the plaintext letters, but the ones that are not yet set all point to -1.
//               (indices are plaintext letters, values are ciphertext).  This is a CUDA array, but isn't
//               listed as such.
//   -cuda_inv_soln_arr: An inverse solution - like the above parameter, but maps ciphertext to plaintext.
//   -cipher_string: The array of integers representing the cipher.
//   -cipher_length: the length of the cipher.
//   -result: an array of length plain_num giving, for each plaintext letter, the best possible probability
//            of a path that assigns that plaintext letter to b, the next ciphertext letter to be fixed.
//    This is basically an array to be changed in place for the return values.
//   -(uni|bi|tri)gram: CUDA arrays representing the unigrams, bigrams, and trigrams.
//   -greenhouse, backpointers: CUDA arrays holding the greenhouse table and the associated backpointers.
//    the greenhouse table holds the probabilities, and the backpointer table holds the back pointers.
//   -tempResult: a table meant to hold the final greenhouse table so it can be refined into the results array.
//    (recall: (1) We can't read directly from a CUDA array, so we have to transfer it. (2) If we pass it in this way,
//    we can reuse it, and so save a hassle with memory management.)
//    -totalSize: The size of the greenhouse table.  We need to have it to transfer between the greenhouse and the 
//               tempResult table.
//
// Returns: Nothing, but alters the results array in place.

void genviterbi(int b, int plain_num, int cipher_num, int * part_soln, int * cuda_inv_soln_arr, int * cipher_string, long cipher_length, float * result, float * unigram, float * bigram, float * trigram, float * greenhouse, int * backpointers, float * tempResult, int totalSize){

// Variables:
//   -i: the index of the column of the greenhouse and backpointer tables that are being filled.
//   -(c|c1|c2): The current / previous / second previous letter in the cipher at index i.
//   -(l|j): loop variables used to iterate over all plaintext letters when filling the result
//           array.
//   -countFnCalls: Used when breaking the CUDA threads into blacks.  Counts the number of blocks
//    used.

  int i;
  int c = 0;
  int c1 = 0;
  int c2 = 0;
  int l;
  int j;
  int countFnCalls = 0;

  // CUDA architecture drops function calls if there more than 512 of them.  
  // This section determines how large the chunks we can use are.
  // We're assuming that the size of the alpohabet (plain_num) in much
  // less than the size of the blocks (threshold).  This works for English,
  // where we're dealing with ~54 at max, but we will likely need to change how
  // the blocks are called when we switch to Hindi (hundreds of characters).

  int threshold = 512;
  int numK = threshold / plain_num;

  // A thread block used to call the CUDA threads.

  dim3 db(plain_num, numK);

  // Fill the first column.

  c = *cipher_string;
  while  (countFnCalls * numK < plain_num){
    rowZero<<< 1, db >>>(greenhouse, plain_num, b, part_soln, cuda_inv_soln_arr, c, unigram, countFnCalls, numK);
    countFnCalls++;
  }

  // Fill the second column.

  countFnCalls = 0;
  c = *(cipher_string + 1);
  while  (countFnCalls * numK < plain_num){
    rowOne<<< 1, db >>>(greenhouse, backpointers, plain_num, b, part_soln, cuda_inv_soln_arr, c, unigram, bigram, countFnCalls, numK);
    countFnCalls++;
  }
  
  // Fill the third column.

  countFnCalls = 0;
  c = *(cipher_string + 2);
  c1 = *(cipher_string + 1);
  c2 = *(cipher_string);
  while  (countFnCalls * numK < plain_num){
    rowTwo<<< 1, db >>>(greenhouse, backpointers, plain_num, b, c, c1, c2, part_soln, cuda_inv_soln_arr, unigram, bigram, trigram, countFnCalls, numK);
    countFnCalls++;
  }  
  
  // Fill the remaining columns.

  for(i = 3; i < cipher_length; i++){
    countFnCalls = 0;
    c = *(cipher_string + i);
    c1 = *(cipher_string + i-1);
    c2 = *(cipher_string + i-2);
    while  (countFnCalls * numK < plain_num){
      rowThreePlus<<< 1, db >>>(greenhouse, backpointers, plain_num, b, c, c1, c2, i, cipher_length, part_soln, cuda_inv_soln_arr, unigram, bigram, trigram, countFnCalls, numK);
      countFnCalls++;
    }  
  }

  // Copy the greenhouse to the tempResult table.

  hipMemcpy(tempResult, greenhouse, (totalSize * sizeof(float)), hipMemcpyDeviceToHost);

  // Fill the result array.

  for(l = 0; l < plain_num; l++){
    *(result + l) = -1;
  }
  for(l = 0; l < plain_num; l++){
    for(j = 0; j < plain_num; j++){
      if(*(tempResult + (((cipher_length - 1) % 3) * plain_num + l) * plain_num + j) > 0){
        if(*(result + j) <= 0){
          *(result + j) = *(tempResult + (((cipher_length - 1) % 3) * plain_num + l) * plain_num + j);
        } else {
          if(*(result + j) > *(tempResult + (((cipher_length - 1) % 3) * plain_num + l) * plain_num + j)){
            *(result + j) = *(tempResult + (((cipher_length - 1) % 3) * plain_num + l) * plain_num + j);
          }
        }
      }
    }
  }
}

/* ---------------------------- End General Functions -------------------------------------- */


/* ------------------------------------ Main------------------------------------------------ */

// The main program.
//
// Calling procedure: If this program is compiled with the name "greenhouse", type
//   >> greenhouse profile
// in the command line, where profile is the name of the profile to be used.
//
//   profiles have the format:
//   line 1: all ciphertext letters, seperated by spaces.
//   line 2: all plaintext letters, seperated by spaces.
//   line 3: the name of the unigram file.
//   line 4: the name of the bigram file.
//   line 5: the name of the trigram file.
//   line 6: the name of the ciphertext file.
//   Current profiles also have a seventh line that gives the
//   location of the plaintext file, which can be used for setup and evaluation
//   purposes.  It won't be read by this program, though.
//
// Note: this program returns a set of stats for each run of the viterbi algorithm, and
//       also the final solution, number of runs per solution size, and the total running time.
//
// e.g., the final lines are of the format:
// 
//Time taken for process: 56:52:38
//
//  Solution: {0:0,1:1,2:3,3:4,4:5,5:6,6:7,7:8,8:9,9:10,10:11,11:12,12:13,13:14,14:15,15:16,16:17,17:18,18:19,19:20,20:21,21:22,22:23,23:24,24:25,25:26,26:27,27:2}
//
//  Solution sizes: {1:1,2:24,3:340,4:938,5:6090,6:27485,7:13285,8:19922,9:10783,10:6764,11:3143,12:202,13:15,14:4,15:1,16:4,17:3,18:2,19:1,20:1,21:1,22:1,23:1,24:1,25:1,26:1,27:1,28:1}
// 
// (this was a large run.)
//
// while the information given in a particular run of the algorithm is of the fromat:
//
//Starting Viterbi Algorithm: 
//
//  Queue size: 418
//
//  Solution size: 3
//
//  Number of passes: 61
//
//  Solution sizes: {1:1,2:24,3:340,4:113}
// 
// Information on the number of zeroed solutions can be obtained by looking at the information given in the different runs of the algorithm.
// Correctness of the solution is not checked by this algorithm.
// Both of these jobs are done by simple python scripts.

int main(int argc, char * argv[]){

  // A generic loop variable.

  int i;

  // This will be the total size of the greenhouse table.

  int totalSize;

  // The array used to read the argument in the commandline function call.

  char * s = "profile";

  // make sure that the calling procedure is correct.

  if (argc != 2){
    //printf("\nUsage: ");
    //printf("\t<insert usage description here>\n\n");
    //exit(0);
  } else {
    s = argv[1];
  }

  // read profile
  ifstream f_profile(s);

  // Lines from the profile.
  // profiles have the format:
  // line 1: all ciphertext letters, seperated by spaces.
  // line 2: all plaintext letters, seperated by spaces.
  // line 3: the name of the unigram file.
  // line 4: the name of the bigram file.
  // line 5: the name of the trigram file.
  // line 6: the name of the ciphertext file.
  // Current profiles also have a seventh line that gives the
  // location of the plaintext file, which can be used for setup and evaluation
  // purposes.  It won't be read by this program, though.

  string cipher_line;
  string plain_line;
  string unigram_name;
  string bigram_name;
  string trigram_name;
  string cipher_name;

  getline(f_profile, cipher_line, '\n');
  getline(f_profile, plain_line, '\n');
  getline(f_profile, unigram_name, '\n');
  getline(f_profile, bigram_name, '\n');
  getline(f_profile, trigram_name, '\n');
  getline(f_profile, cipher_name, '\n');
  
  f_profile.close();

  // We don't know the sizes of the alphabets beforehand, so we're reading them into 
  // vectors.  Recall that they're only being used for setup: in the actual algorithm, 
  // we'll only use numbers.
  //
  // Use the cipher and plaintext lines to create the corresponding lists:

  for(i = 0; i < cipher_line.size(); i++){
    if(cipher_line[i] != ' '){
      cipher_alpha.push_back(cipher_line[i]);
    }
  }
  cipher_num = cipher_alpha.size();
  for(i = 0; i < cipher_num; i++){
    inv_cipher[cipher_alpha[i]] = i;
  }
  for(i = 0; i < plain_line.size(); i++){
    if(plain_line[i] != ' '){
      plain_alpha.push_back(plain_line[i]);
    }
  }
  plain_num = plain_alpha.size();
  for(i = 0; i < plain_num; i++){
    inv_plain[plain_alpha[i]] = i;
  }

  // Read the unigrams, bigrams, and trigrams

  // for the unigrams, read each line, process it into its parts,
  // and add each entry into a unigram map.
  // The basic map has size plain_num, with a default entry of -1.
  // unigram is an array whose ith index is *(unigram + i)

  unigram = (float *) malloc(plain_num * sizeof(float));
  for(i = 0; i < plain_num; i++){
    *(unigram + i) = -1;
  }
  ifstream unigrams_file(unigram_name.c_str());
  string temp;
  float uni_total = 0;
  while(getline(unigrams_file, temp, '\n')){
    char char_1;
    i = 0;
    while((temp[i] == ' ') || (temp[i] == '\t')){
      i++;
    }
    char_1 = temp[i];
    i++;
    while((temp[i] == ' ') || (temp[i] == '\t')){
      i++;
    }
    string str = "";
    while((temp[i] != ' ') && (temp[i] != '\t')){
      str.push_back(temp[i]);
      i++;
    }
    float uni_prob = atof(str.c_str());
    *(unigram + inv_plain[char_1]) = -1 * log(uni_prob);
    uni_total += uni_prob;
  }
  for(i = 0; i < plain_num; i++){
     if((*(unigram + i) != -1) && isfinite(*(unigram + i))){
       *(unigram + i) += log(uni_total);
     } else {
       *(unigram + i) = -1;
     }
  }
  unigrams_file.close();

  // Read the bigram file.
  // This is an array of size plain_num^2, with default values -1.
  // the index for i following j is *(bigram + plain_num * j + i).

  int p_2 = plain_num * plain_num;
  bigram = (float *) malloc(p_2 * sizeof(float));
  for(i = 0; i < p_2; i++){
    *(bigram + i) = -1;
  }
  ifstream bigrams_file(bigram_name.c_str());
  float bi_total = 0;
  while(getline(bigrams_file, temp, '\n')){
    char char_1;
    char char_2;
    i = 0;
    while((temp[i] == ' ') || (temp[i] == '\t')){
      i++;
    }
    char_1 = temp[i];
    i++;
    while((temp[i] == ' ') || (temp[i] == '\t')){
      i++;
    }
    char_2 = temp[i];
    i++;
    while((temp[i] == ' ') || (temp[i] == '\t')){
      i++;
    }
    string str = "";
    while((temp[i] != ' ') && (temp[i] != '\t')){
      str.push_back(temp[i]);
      i++;
    }
    float bi_prob = atof(str.c_str());
    *(bigram + inv_plain[char_2] + (inv_plain[char_1] * plain_num)) = -1 * log(bi_prob);
    bi_total += bi_prob;
  }
  for(i = 0; i < p_2; i++){
     if((*(bigram + i) != -1) && isfinite(*(bigram + i))){
       // TODO: I think it shouldn't normalize like this, but experiments showed better results with this
       *(bigram + i) += log(bi_total);
     } else {
       *(bigram + i) = -1;
     }
  }
  bigrams_file.close();

  // Read the trigram file.
  // This is an array of size plain_num^3, with default values -1.
  // the index for i following j following k is *(trigram + plain_num * (plain_num * k + j) + i).

  int p_3 = plain_num * plain_num * plain_num;
  trigram = (float *) malloc(p_3 * sizeof(float));
  for(i = 0; i < p_3; i++){
    *(trigram + i) = -1;
  }
  ifstream trigrams_file(trigram_name.c_str());
  float tri_total = 0;
  while(getline(trigrams_file, temp, '\n')){
    char char_1;
    char char_2;
    char char_3;
    i = 0;
    while((temp[i] == ' ') || (temp[i] == '\t')){
      i++;
    }
    char_1 = temp[i];
    i++;
    while((temp[i] == ' ') || (temp[i] == '\t')){
      i++;
    }
    char_2 = temp[i];
    i++;
    while((temp[i] == ' ') || (temp[i] == '\t')){
      i++;
    }
    char_3 = temp[i];
    i++;
    while((temp[i] == ' ') || (temp[i] == '\t')){
      i++;
    }
    string str = "";
    while((temp[i] != ' ') && (temp[i] != '\t')){
      str.push_back(temp[i]);
      i++;
    }
    float tri_prob = atof(str.c_str());
    *(trigram + inv_plain[char_3] + (inv_plain[char_2] * plain_num) + (inv_plain[char_1] * p_2)) = -1 * log(tri_prob);
    tri_total += tri_prob;
  }
  for(i = 0; i < p_3; i++){
     if((*(trigram + i) != -1) && isfinite(*(trigram + i))){
       // TODO: I think it shouldn't normalize like this, but experiments showed better results with this
       *(trigram + i) += log(tri_total);
     } else {
       *(trigram + i) = -1;
     }
  }
  trigrams_file.close();

  // Create the cipher string.
  // Recall that we're going to turn everything into numbers, and first read everything into a vector,
  // the transfer the vector to an array.

  ifstream cipher_file(cipher_name.c_str());
  while( getline(cipher_file, temp, '\n')){
    for(i = 0; i < temp.size(); i++){
      if(temp[i] != ' '){
        cipher_string_vec.push_back(inv_cipher[temp[i]]);
      }
    }
  }
  cipher_length = cipher_string_vec.size();
  cipher_string = (int *) malloc(cipher_length * sizeof(int));
  for(i = 0; i < cipher_length; i++){
    *(cipher_string + i) = cipher_string_vec[i];
  }
  cipher_file.close();

  // get the locations of the last characters in the cipher.
  // This will be used to determine the order in which letters are 
  // added to the solution.

  int num_last = 0;
  map<int, int> last;
  map<int, int> lastcount;
  for(i = cipher_length - 1; i >= 0; i--){
    if(last.count(*(cipher_string + i)) == 0){
      last[*(cipher_string + i)] = i;
      lastcount[num_last] = i;
      num_last++;
    }
  }

  // Set up the table in the GPU, and copy out the unigrams, bigrams, and trigrams into their own tables here.
  // Put the backpointers table there, too.

  totalSize = cipher_num * plain_num * plain_num * 3;

  int * test_array;
  hipMalloc((void**)&test_array, (plain_num * sizeof(int)));

  float * result = (float *) malloc(plain_num * sizeof(float));
  float * tempResult = (float *) malloc(totalSize * sizeof(float));

  // Create the greenhouse array.

  hipMalloc((void**)&greenhouse, (totalSize * sizeof(float)));
  hipMalloc((void**)&backpointers, (totalSize * sizeof(int)));

  hipMemcpy(greenhouse, tempResult, (totalSize * sizeof(float)), hipMemcpyHostToDevice);

  // Copy the unigrams, bigrams, and trigrams into the GPU.

  hipMalloc((void**)&cudaUnigrams, (plain_num * sizeof(float)));
  hipMemcpy(cudaUnigrams, unigram, (plain_num * sizeof(float)), hipMemcpyHostToDevice);

  hipMalloc((void**)&cudaBigrams, (plain_num * plain_num * sizeof(float)));
  hipMemcpy(cudaBigrams, bigram, (plain_num * plain_num * sizeof(float)), hipMemcpyHostToDevice);

  hipMalloc((void**)&cudaTrigrams, (plain_num * plain_num * plain_num * sizeof(float)));
  hipMemcpy(cudaTrigrams, trigram, (plain_num * plain_num * plain_num * sizeof(float)), hipMemcpyHostToDevice);

  // Make sure the part_soln and part_inv are in the GPU here.

  int * inv_soln_arr = (int *) malloc(cipher_num*sizeof(int));  
  int * curr_soln_arr = (int *) malloc(plain_num * sizeof(int));
  int * cuda_curr_soln_arr;
  hipMalloc((void**)&cuda_curr_soln_arr, (plain_num * sizeof(int)));
  int * cuda_inv_soln_arr;
  hipMalloc((void**)&cuda_inv_soln_arr, (cipher_num * sizeof(int)));

  // Some stuff to get the GPU to work the way we want.

  hipGetTextureReference(&texRefPtr, HIP_SYMBOL("texRef"));
  texRef.addressMode[0] = hipAddressModeWrap;
  texRef.addressMode[1] = hipAddressModeWrap;
  texRef.filterMode     = hipFilterModeLinear;
  texRef.normalized     = true;

  // set up priority heap.
  // set up a map of sizes.
  // Set up a starting solution and add it to the heap.

  map<int, int> start_soln;

  // Normally, the solution is empty. here.
  // The following line adds the restriction that spaces map to spaces
  // to the solution (the ciphers we're using always end in a space).
  // Comment or uncomment this line as needed.

  start_soln[0] = 0;
  map<int, int> soln_sizes;
  soln_sizes[start_soln.size()] = 1;
  long pass_num = 1;
  
  // A boolean variable that will tell us when we're finished looking for the solution.

  bool found = false;

  // prime the priority queue with the first solution.

  int l;
  time_t start_time = time (NULL);   
  priority_queue< pair<float, map<int, int> >, vector<pair<float, map<int, int> > >, pair_compare > aStar;
  aStar.push(pair<float, map<int, int> >(-1, start_soln));

  // The main part of the program: pop solutions and run the genviterbi algorithm to grow solutions until
  // the final solution is found.

  while(!(aStar.empty()) && !found){

    // pop the best solution, and set up the next run.

    map<int, int> curr_soln = aStar.top().second;
    float curr_soln_prob = aStar.top().first;
    for(i = 0; i < plain_num; i++){  // Copy all curr soln to curr_soln_arr
      if(curr_soln.count(i) > 0){
        *(curr_soln_arr + i) = curr_soln[i];
      } else {
        *(curr_soln_arr + i) = -1;
      }
    }
    aStar.pop();

    // print some stats.

    cout << "Starting Viterbi Algorithm: \n" << endl;
    cout << "  Queue size: " << (aStar.size() + 1) << "\n" << endl;
    cout << "  Solution size: " << curr_soln.size() << "\n" << endl;
    cout << "  Solution probability " << curr_soln_prob << "\n" << endl;
    cout << "  Number of passes: " << pass_num++ << "\n" << endl;
    
    stringstream temp_soln;
    map<int, int>::iterator soln_iter;
    for(soln_iter = soln_sizes.begin(); soln_iter != soln_sizes.end(); ++soln_iter){
      temp_soln << "," <<  soln_iter->first << ":" << soln_iter->second;
    }
    string soln_string = temp_soln.str();
    soln_string.replace(0, 1, "{");
    soln_string += "}";
    cout << "  Solution sizes: " << soln_string << "\n" << endl;

    // Check to see if we're done (the solution is large enough to cover all ciphertext letters).
    // If so, set the solution to the current solution and exit the loop.
    // Otherwise, find the next letter to be added.

    if(curr_soln.size() >= cipher_num){
      found = true;
      start_soln = curr_soln;
    } else {
      int curr_endpoint = *(cipher_string + lastcount[curr_soln.size()]);

      // This part will create the inverse partial solution.

      for(i = 0; i < cipher_num; i++){
        *(inv_soln_arr + i) = -1;
      }
      for(i = 0; i < plain_num; i++){
        if(*(curr_soln_arr + i) >= 0){
          *(inv_soln_arr + *(curr_soln_arr + i)) = i;
        }
      }

      // Copy the partial solution and the inverse partial solution to the CUDA arrays.

      hipMemcpy(cuda_curr_soln_arr, curr_soln_arr, (plain_num * sizeof(int)), hipMemcpyHostToDevice);
      hipMemcpy(cuda_inv_soln_arr, inv_soln_arr, (plain_num * sizeof(int)), hipMemcpyHostToDevice);

      // run the actual genviterbi algorithm.

      genviterbi(curr_endpoint, plain_num, cipher_num, cuda_curr_soln_arr, cuda_inv_soln_arr, cipher_string, cipher_length, result, cudaUnigrams, cudaBigrams, cudaTrigrams, greenhouse, backpointers, tempResult, totalSize);

      // For each possible extension to the solution, format the next partial solution
      // and add it to the heap.

      for(l = 0; l < plain_num; l++){
        if(*(result + l) >= 0){
          map<int, int> next_soln;
          map<int, int>::iterator curr_iter;
          for(curr_iter = curr_soln.begin(); curr_iter != curr_soln.end(); ++curr_iter){
            next_soln[curr_iter->first] = curr_iter->second;
          }
          next_soln[l] = curr_endpoint;
          float next_prob = *(result + l);
          if(next_prob > 0){
            aStar.push(pair<float, map<int, int> >(next_prob, next_soln));
            int size_count = 1;
            if(soln_sizes.count(curr_soln.size() + 1) > 0){
              size_count = soln_sizes[curr_soln.size() + 1] + 1;
            }
            soln_sizes[curr_soln.size() + 1] = size_count;
          }
        }
      }
    }

  }

  // return the results.

  time_t stop_time = time (NULL);
  long total_time = stop_time - start_time;
  int seconds = total_time % 60;
  int minutes = ((total_time - seconds) / 60) % 60;
  long hours = (((total_time - seconds) / 60) - minutes) / 60;
  cout << "Time taken for process: " << hours << ":" << minutes << ":" << seconds << "\n"<< endl;
  if(start_soln.size() == cipher_num){
    stringstream soln_stream_1;
    map<int, int>::iterator soln_iter;
    for(soln_iter = start_soln.begin(); soln_iter != start_soln.end(); ++soln_iter){
      soln_stream_1 << "," <<  soln_iter->first << ":" << soln_iter->second;
    }
    string soln_string = soln_stream_1.str();
    soln_string.replace(0, 1, "{");
    soln_string += "}";
    cout << "  Solution: " << soln_string << "\n" << endl;

    stringstream soln_stream_2;
    for(soln_iter = soln_sizes.begin(); soln_iter != soln_sizes.end(); ++soln_iter){
      soln_stream_2 << "," <<  soln_iter->first << ":" << soln_iter->second;
    }
    soln_string = soln_stream_2.str();
    soln_string.replace(0, 1, "{");
    soln_string += "}";
    cout << "  Solution sizes: " << soln_string << "\n" << endl;
  } else {
    cout << "Warning: could not complete solution." << endl;
  }

  // free up everything.
  free(curr_soln_arr);
  free(result);
  free(tempResult);
  free(unigram);
  free(bigram);
  free(trigram);
  free(cipher_string);
  free(inv_soln_arr);

  hipFree(greenhouse);
  hipFree(backpointers);
  hipFree(cudaUnigrams);
  hipFree(cudaBigrams);
  hipFree(cudaTrigrams);
  hipFree(cuda_curr_soln_arr);
  hipFree(cuda_inv_soln_arr);
  hipFree(test_array);

  return 0;
}

/* -------------------------------- End Main------------------------------------------------ */
